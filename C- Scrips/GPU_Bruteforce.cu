/*
CUDA code for GPU optimization of camera placement problem
With support of 2-coverage Region of Interest
Author : Vegard Tveit
Date : 17.04.2018
Comment : The user has to specify: 

	- Number of sensors to be placed
	- Number of possible combinations(nchoosek)
	- Modify UNISIZE 
	- Number of datapoints
	- Number of possible placement points
	- Number of possible pan angles
	- "subsets.txt", "annotations.txt" and "combinations.txt" 

*/
// Initial Setup

#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <vector>

#include <new>
#define UNISIZE 9084
#include <cmath>
#include <algorithm>
#include <numeric>
#include <functional>
#include <fstream>
__global__ void mykernel(int* annotations, int* devarr, bool* subs, int* sum, unsigned long len, unsigned long nsubs, unsigned long usize, int roisum)
{		
	// Kernel function to run on GPU
	// Defining variables (stored in each kernel)
	unsigned long th_id = blockIdx.x * blockDim.x + threadIdx.x;		// The id of the current thread	 	
	int barr[9084] = {0}; //Array for storing coverage
	int totsum = 0; // Sum of covered points	
	int count_roi = 0; 
	int penalty = 0; 
	int alpha = 4000; 
	
	int ct = 0; 
	if(th_id < len){
		for(unsigned long i = 0; i < nsubs; i++)
		{		
			int ind = devarr[th_id*nsubs + i];
			for(unsigned long j = 0; j < usize; j++)
			{
				if(subs[ind*usize + j]){
					barr[j] += 1; 
				
				}
			}
		
		}			
		for(int i = 0 ; i < usize ; i++){
			if(barr[i] >= annotations[i]){
				totsum += 1; 
				if(annotations[i] == 2){
					count_roi += 1; 
				}
			}
		}
		if(count_roi > 0 && count_roi < roisum){
			penalty = alpha*(roisum/count_roi); 
		}
		if(count_roi == 0){
			penalty = alpha*roisum; 
		}
		
		sum[th_id] = totsum - penalty;			
	}else sum[th_id] = 0; 
	
}

void readfromtxt(){

	//int num_sensors = 1; 
	//int ncombs = 210; 
	int num_sensors = 5; 
	int ncombs = 1.5e8;
	 
	//unsigned long num_sensors = 3; 
	//unsigned long ncombs = 1521520;
	
	//unsigned long num_sensors = 4; 
	//unsigned long ncombs = 78738660;

	unsigned long ndp = 9084; 
	unsigned long campos = 83; 
	unsigned long numpans = 3; 
	
	std::cout << "num combs : " << ncombs << std::endl; 
	
	// Dynamically allocate arrays on CPU
	int* array = (int*)malloc(ncombs*num_sensors*sizeof(int)); 	
	bool* subs_array = (bool*)malloc(ndp*campos*numpans*sizeof(bool*));
	int* annot_array = (int*)malloc(ndp*sizeof(int));
	
	//Load subsets from txt file and store in 1D array
	std::ifstream subsfile("Subsets_4cam.txt"); 
	double b;
	unsigned long col_s = 0; 
	while (subsfile >> b)
	{
		subs_array[col_s] = (bool) b;
		
		col_s +=1; 		
		
	}
	for(int i = 0; i < 15 ; i++){
		std::cout << subs_array[i] << std::endl;
	}
	std::cout << std::endl << std::endl  <<std::endl; 
	std::cout << col_s << " is col_s" << std::endl; 
	// Store combinations array in a 1D array
	std::ifstream myfile("combtests_1.txt"); 
	double bb; 
	unsigned long col = 0; 
	std::cout << "Hei!" << std::endl; 
	while (myfile >> bb)
   	 {	
		array[col] = (int) bb;		
		if(col < 10) std::cout << bb << std::endl; 
		col += 1;  
   	 }	
	
			
	std::cout << col << " is col" << std::endl; 
	// Store annotations in a 1D array
	// The annotation of a point describes whether it is
	// a ROI, obstacle or normal data point
	std::ifstream annotfile("Annotations.txt"); 
	double an;
	unsigned long col2 = 0; 

	while (annotfile >> an)
    {
		annot_array[col2] =(int) an;		
		col2 += 1; 
    }	
	
	// Make annotation array (to be used inside kernel)
	int* init_cov = (int*)malloc(ndp*sizeof(int)); 
	//int init_cov[ndp];
	int c = 0; 
	for(int i = 0 ; i < ndp ; i++){
		if(annot_array[i] == 2){
			c += 1; 
			init_cov[i] = 2; 
		}else{
			init_cov[i] = 1; 
		}
	}
	std::cout << "Num of roi dp: " << c << std::endl; 
	

	//GPU variables
	unsigned long n_threads_per_block = 1024; 
	unsigned long n_blocks = (ncombs + n_threads_per_block - 1)/n_threads_per_block;
	
	std::cout << "Number of blocks :" << n_blocks << std::endl; 
	unsigned long data_n = n_blocks*n_threads_per_block; // Total number of available threads
	
	//Vectorize array for GPU calculations
	unsigned long chop_combs;	
	chop_combs = ncombs; 
	
	std::cout << "No. of available threads: " <<  data_n << std::endl; 
	std::cout << "Number of used threads : " << chop_combs << std::endl;
		
	size_t i_datasize = chop_combs*sizeof(int); 
	size_t array_datas = chop_combs*num_sensors*sizeof(int); 
	size_t bool_subs_size = ndp*numpans*campos*sizeof(bool); 
	size_t annot_size = ndp*sizeof(int);
	
	std::cout << "i_datasize [bytes] : " << i_datasize <<  std::endl; 
	
	// Allocate CPU Memory
	int* sum_host = new int[chop_combs]; 
	
	std:: cout << "Array size : " << array_datas <<" and subs size " << bool_subs_size << std::endl; 
	
	// Allocate GPU Memory
	int* annot_dev;
	bool* subs_dev; 
	int* sum_dev; 
	int* array_dev;
	
	hipMalloc(&subs_dev,bool_subs_size);
	hipMalloc(&array_dev, array_datas);
	hipMalloc(&sum_dev,i_datasize); 
	hipMalloc(&annot_dev,annot_size); 
	
	// Copy host (CPU) arrays to device (GPU) arrays
	hipMemcpy(subs_dev, subs_array, bool_subs_size, hipMemcpyHostToDevice);	
	hipMemcpy(sum_dev, sum_host, i_datasize, hipMemcpyHostToDevice);	
	hipMemcpy(array_dev, array, array_datas, hipMemcpyHostToDevice);	
	hipMemcpy(annot_dev,init_cov,annot_size,hipMemcpyHostToDevice);
	
	// Run "mykernel" function on GPU threads with gpu timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);	
	hipEventRecord(start);
	
	mykernel <<< n_blocks,n_threads_per_block >>> (annot_dev,array_dev,subs_dev,sum_dev,chop_combs,num_sensors,ndp,c);
	
	hipDeviceSynchronize();
	hipEventRecord(stop);	
	hipEventSynchronize(stop);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("The elapsed time for kernel execution was %.2f ms\n", milliseconds);
	// Copy results back to cpu memory
	hipMemcpy(sum_host, sum_dev, i_datasize, hipMemcpyDeviceToHost);
	
	// Post process	
	int max = 0; 
	unsigned long  ind = 0; 
	for (unsigned long  i = 0; i < chop_combs ; i++){
		if(sum_host[i] > max){
			max = sum_host[i];
			ind = i; 
		} 
	}
	std::cout << "Max val : " << max << std::endl; 	
	printf("Highest coverage value at index %lu. \n",ind); 
	std::cout << "The index represents camera index: ";  
	for(int m = 0; m < num_sensors ; m++){
		
		printf("%i ", array[ind*num_sensors + m]);
	}
	std::cout << std::endl; 
	
	//Free allocated memory on CPU and GPU
	hipFree(subs_dev);
	hipFree(sum_dev); 
	hipFree(array_dev);
	delete[] sum_host; 
	free(array); 
	free(subs_array);
	
}


